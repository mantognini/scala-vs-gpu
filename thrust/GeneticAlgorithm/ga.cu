#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>

typedef float Real;

__host__ __device__
bool isClose(Real value, Real target, Real flex)
{
    return (1 - flex) * target <= value && value <= (1 + flex) * target;
}

template <typename T, typename U>
struct SumPair {
    typedef typename thrust::pair<T, U> Pair;

    SumPair() {
    }

    __host__ __device__
    Pair operator()(Pair const& as, Pair const& bs) const {
        return Pair(as.first + bs.first, as.second + bs.second);
    }
};

struct Settings {
    Settings(unsigned int size, unsigned int K, unsigned int M, unsigned int N, unsigned int CO)
        : size(size)
        , K(K)
        , M(M)
        , N(N)
        , CO(CO) {
        if (!isValid()) {
            throw new std::domain_error("Invalid settings");
        }
    }

    const unsigned int size; ///< population size
    const unsigned int K; ///< number of killed per generation
    const unsigned int M; ///< number of mutated per generation
    const unsigned int N; ///< number of new individuals (random) per generation
    const unsigned int CO; ///< number of new individuals (cross over) per generation

    /// Make sure the settings are valid
    bool isValid() const {
        // K, M < size
        if (K >= size || M >= size) {
            return false;
        }

        // N + CO = K
        if (N + CO != K) {
            return false;
        }

        return true;
    }
};


class Population
{
public:
    // Type Aliases

    // Define Entity & Fitness Pop using SoA (Structure of Arrays)
    typedef thrust::pair<Real, Real> Params;
    typedef thrust::device_vector<Params> EntityPopDevice;
    typedef thrust::device_vector<Real> FitnessPopDevice;
    typedef thrust::host_vector<Params> EntityPopHost;
    typedef thrust::host_vector<Real> FitnessPopHost;


    // Equation :
    //
    // Sin[x - 15] / x * (y - 7) (y - 30) (y - 50) (x - 15) (x - 45)
    //
    // Range : (x, y) in [9, 100] x [7, 50]

public:
    // Public API

    /*!
     * Ctor
     *
     * @param settings settings for the algorithm
     */
    Population(Settings settings)
        : settings(settings) {
    }

    /// Apply the genetic algorithm until the population stabilise and return the best entity
    Params run() {
        // Use a counter for random number so that the random number are really random !
        thrust::counting_iterator<std::size_t> randomCount(0); // (for generator only)

        // Step 1 + 2.
        // -----------
        //
        // Generate a population & evaluate it
        EntityPopDevice epopd(settings.size);
        FitnessPopDevice fpopd(settings.size);
        thrust::transform(randomCount, randomCount + settings.size, epopd.begin(), generator);
        randomCount += settings.size;
        // Evaluate it
        thrust::transform(epopd.begin(), epopd.end(), fpopd.begin(), evaluator);
        // Now sort it
        thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

        // Random generators
        thrust::default_random_engine rng;

        unsigned int rounds = 0;

        do {
            ++rounds;

            // Step 3.
            // -------
            //
            // Remove the worse K individuals

            // Skipped -> replace those entities with step 5 & 6


            // Step 4.
            // -------
            //
            // Mutate M individuals of the population

            // Choose M random individuals from the living ones, that is in range [0, size-K[

            for (unsigned int count = 0; count < settings.M; ++count) {
                const unsigned int rangeStart = 0;
                const unsigned int rangeEnd = settings.size - settings.K - 1;
                thrust::uniform_int_distribution<unsigned int> uniform(rangeStart, rangeEnd);
                const unsigned int index = uniform(rng);

                // mutate the entity and recompute its fitness
                Params ps = mutator(epopd[index]);
                epopd[index] = ps;
                fpopd[index] = evaluator(ps);
            }


            // Step 5.
            // -------
            //
            // Create CO new individuals with CrossOver

            // Replace the last CO entities before the N last ones (see comment at step 3)
            for (unsigned int i = settings.size - settings.N - 1, count = 0; count < settings.CO; ++count) {
                // Select two random entities from the living ones, that is in range [0, size-K[
                const unsigned int rangeStart = 0;
                const unsigned int rangeEnd = settings.size - settings.K - 1;
                thrust::uniform_int_distribution<unsigned int> uniform(rangeStart, rangeEnd);
                const unsigned int first = uniform(rng);
                const unsigned int second = uniform(rng);

                Params ps = crossover(epopd[first], epopd[second]);
                epopd[i] = ps;
                fpopd[i] = evaluator(ps);
            }


            // Step 6.
            // -------
            //
            // Generate N new individuals randomly

            // Replace the last N entities (see comment at step 3)
            thrust::transform(randomCount, randomCount + settings.N, epopd.end() - settings.N - 1, generator);
            randomCount += settings.N;
            // Evaluate it
            thrust::transform(epopd.end() - settings.N - 1, epopd.end(),
                              fpopd.begin()  - settings.N - 1,
                              evaluator);


            // Step 7.
            // -------
            //
            // Evaluate the current population

            // The evaluation of new entities was already done in step 3 to 6
            // So we only sort the population

            // Sort the data
            thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());


            // Step 8.
            // -------
            //
            // Goto Step 3 if the population is not stable yet

        } while (!terminator(epopd));

        std::cout << "#rounds = " << rounds << std::endl;

        // Step 9.
        // -------
        //
        // Identify the best individual from the current population

        return epopd.front(); // the population is already sorted;
    }

// private:
    // Private API
    // But public to work with thrust / cuda ...

    static const Real MIN_X = 9, MAX_X = 100, MIN_Y = 7, MAX_Y = 50;

    // Generator; random parameters in [MIN_X, MAX_X] x [MIN_Y, MAX_Y]
    struct Generator {
        Generator()
            : rng(std::time(0))
            , distX(MIN_X, MAX_X)
            , distY(MIN_Y, MAX_Y) {
        }

        __host__ __device__
        Params operator()(std::size_t n) { // The n is used to drop some random numbers
            rng.discard(2 * n); // since we take two random numbers
            return Params(distX(rng), distY(rng));
        }

    private:
        // Random generators
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<Real> distX, distY;
    } generator;

    // Evaluator; the biggest the better
    struct Evaluator {
        __host__ __device__
        Real operator()(Params const& ps) {
            Real x = ps.first, y = ps.second;

            return std::sin(x - 15) / x * (y - 7) * (y - 30) * (y - 50) * (x - 15) * (x - 45);
        }
    } evaluator;

    // CrossOver; takes the average of the two entities
    __host__ __device__
    Params crossover(Params const& as, Params const& bs) {
        Real ax = as.first,
             ay = as.second,
             bx = bs.first,
             by = bs.second;

        return Params((ax + bx) / Real(2), (ay + by) / Real(2));
    }


    // Mutator; takes a normal distribution to shift the current value
    __host__ __device__
    Params mutator(Params const& ps) {
        // TODO implement me !
        return ps;
    }

    struct IsOut {
        IsOut(Real avgX, Real avgY, Real epsilon)
            : avgX(avgX)
            , avgY(avgY)
            , epsilon(epsilon) {
        }

        __host__ __device__
        bool operator()(Params const& ps) const {
            return !isClose(ps.first, avgX, epsilon) || !isClose(ps.second, avgY, epsilon);
        }

        const Real avgX, avgY, epsilon;
    };

    // Terminator; stop evolution when population has (relatively) converged
    bool terminator(EntityPopDevice const& pop) {
        // Compute average on x and y axes
        const SumPair<Real, Real> reducer;
        Params sum = thrust::reduce(pop.begin(), pop.end(), Params(0, 0), reducer);
        Real avgX = sum.first / pop.size();
        Real avgY = sum.second / pop.size();

        // Stop when 75% of the population is in the range [(1 - ε) * µ, (1 + ε) * µ]
        const std::size_t maxOuts = pop.size() * 0.25;
        const Real EPSILON = 0.02;

        const IsOut predicate(avgX, avgY, EPSILON);
        const std::size_t outs = thrust::count_if(pop.begin(), pop.end(), predicate);

        return outs <= maxOuts;
    }

private:
    // Data
    Settings settings;
};


struct Action {
    Action(Population& popref)
        : popref(popref) {
    }

    Population::Params operator()() const {
        return popref.run();
    }

    std::string csvdescription() const {
        return "ø"; // no explicit parameters for the computation
    }

    Population& popref;
};

std::ostream& operator<<(std::ostream& out, Population::Params const& ps)
{
    return out << ps.first << "," << ps.second;
}

#include "stats.hpp"

int main(int, char const**)
{
    // Settings
    const Settings settings(1000, 100, 50, 50, 50);

    // Create the population
    Population pop(settings);

    // Run the Genetic Algorithm
    stats<Action, Population::Params>(Action(pop), 100);

    return 0;
}


