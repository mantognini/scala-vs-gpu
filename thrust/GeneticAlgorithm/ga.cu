#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include "stats.hpp"

typedef float Real;

bool isClose(Real value, Real target, Real flex)
{
    return (1 - flex) * target <= value && value <= (1 + flex) * target;
}

struct Settings {
    Settings(unsigned int size, unsigned int K, unsigned int M, unsigned int N, unsigned int CO)
        : size(size)
        , K(K)
        , M(M)
        , N(N)
        , CO(CO) {
        if (!isValid()) {
            throw new std::domain_error("Invalid settings");
        }
    }

    const unsigned int size; ///< population size
    const unsigned int K; ///< number of killed per generation
    const unsigned int M; ///< number of mutated per generation
    const unsigned int N; ///< number of new individuals (random) per generation
    const unsigned int CO; ///< number of new indifiduals (cross over) per generation

    /// Make sure the settings are valid
    bool isValid() const {
        // K, M < size
        if (K >= size || M >= size) {
            return false;
        }

        // N + CO = K
        if (N + CO != K) {
            return false;
        }

        return true;
    }
};


class Population
{
public:
    // Type Aliases

    // Define Entity & Fitness Pop using SoA (Structure of Arrays)
    typedef thrust::pair<Real, Real> Params;
    typedef thrust::device_vector<Params> EntityPopDevice;
    typedef thrust::device_vector<Real> FitnessPopDevice;
    typedef thrust::host_vector<Params> EntityPopHost;
    typedef thrust::host_vector<Real> FitnessPopHost;


    // Equation :
    //
    // Sin[x - 15] / x * (y - 7) (y - 30) (y - 50) (x - 15) (x - 45)
    //
    // Range : (x, y) in [9, 100] x [7, 50]

public:
    // Public API

    /*!
     * Ctor
     *
     * @param settings settings for the algorithm
     */
    Population(Settings settings)
        : settings(settings) {
    }

    /// Apply the genetic algorithm until the population stabilise and return the best entity
    Params run() {

        // Step 1 + 2.
        // -----------
        //
        // Generate a population & evaluate it
        EntityPopDevice epopd(settings.size);
        FitnessPopDevice fpopd(settings.size);
        thrust::generate(epopd.begin(), epopd.end(), generator);
        // Evaluate it
        thrust::transform(epopd.begin(), epopd.end(), fpopd.begin(), evaluator);
        // Now sort it
        thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

        // Copy data back to host
        EntityPopHost epoph = epopd;
        FitnessPopHost fpoph = fpopd;

        do {
            // Step 3.
            // -------
            //
            // Remove the worse K individuals

            // Skipped -> replace those entities with step 5 & 6


            // Step 4.
            // -------
            //
            // Mutate M individuals of the population

            // Choose M random individuals from the living ones, that is in range [0, size-K[

            // TODO implement me !


            // Step 5.
            // -------
            //
            // Create CO new individuals with CrossOver

            // Replace the last CO entities before the N last ones (see comment at step 3)

            // TODO implement me !


            // Step 6.
            // -------
            //
            // Generate N new individuals randomly

            // Replace the last N entities (see comment at step 3)

            // TODO implement me !


            // Step 7.
            // -------
            //
            // Evaluate the current population

            // The evaluation of new entities was already done in step 3 to 6
            // So we only sort the population

            // TODO implement me !


            // Step 8.
            // -------
            //
            // Goto Step 3 if the population is not stable yet

        } while (!terminator(epoph));

        // Step 9.
        // -------
        //
        // Identify the best individual from the current population

        // TODO implement me !
        return Params();
    }

// private:
    // Private API
    // But public to work with thrust / cuda ...

    static const Real MIN_X = 9, MAX_X = 100, MIN_Y = 7, MAX_Y = 50;

    // Generator; random parameters in [MIN_X, MAX_X] x [MIN_Y, MAX_Y]
    struct Generator {
        Generator()
            :rng(std::rand())
            , distX(MIN_X, MAX_X)
            , distY(MIN_Y, MAX_Y) {
        }

        __host__ __device__
        Params operator()() {
            return Params(distX(rng), distY(rng));
        }

    private:
        // Random generators
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<Real> distX, distY;
    } generator;

    // Evaluator; the biggest the better
    struct Evaluator {
        __host__ __device__
        Real operator()(Params const& ps) {
            Real x = ps.first, y = ps.second;

            return std::sin(x - 15) / x * (y - 7) * (y - 30) * (y - 50) * (x - 15) * (x - 45);
        }
    } evaluator;

    // CrossOver; takes the average of the two entities
    __host__ __device__
    Params crossover(Params const& as, Params const& bs) {
        // TODO implement me !
        return Params();
    }


    // Mutator; takes a normal distribution to shift the current value
    __host__ __device__
    Params mutator(Params const& ps) {
        // TODO implement me !
        return ps;
    }


    // Terminator; stop evolution when population has (relatively) converged
    bool terminator(EntityPopHost const& pop) {
        // TODO implement me !
        return true;
    }

private:
    // Data
    Settings settings;
};


struct Action {
    Action(Population& popref)
        : popref(popref) {
    }

    Population::Params operator()() const {
        return popref.run();
    }

    std::string csvdescription() const {
        return "ø"; // no explicit parameters for the computation
    }

    Population& popref;
};

std::ostream& operator<<(std::ostream& out, Population::Params const& ps)
{
    return out << ps.first << "," << ps.second;
}

#include "stats.hpp"

int main(int, char const**)
{
    // Settings
    const Settings settings(1000, 100, 50, 50, 50);

    // Create the population
    Population pop(settings);

    // Run the Genetic Algorithm
    stats<Action, Population::Params>(Action(pop), 100);

    return 0;
}


