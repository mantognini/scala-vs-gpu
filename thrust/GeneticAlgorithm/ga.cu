#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>

typedef float Real;

__host__ __device__
bool isClose(Real value, Real target, Real flex)
{
    return (1 - flex) * target <= value && value <= (1 + flex) * target;
}

struct Settings {
    Settings(unsigned int size, unsigned int K, unsigned int M, unsigned int N, unsigned int CO)
        : size(size)
        , K(K)
        , M(M)
        , N(N)
        , CO(CO) {
        if (!isValid()) {
            throw new std::domain_error("Invalid settings");
        }
    }

    const unsigned int size; ///< population size
    const unsigned int K; ///< number of killed per generation
    const unsigned int M; ///< number of mutated per generation
    const unsigned int N; ///< number of new individuals (random) per generation
    const unsigned int CO; ///< number of new individuals (cross over) per generation

    /// Make sure the settings are valid
    bool isValid() const {
        // K, M < size
        if (K >= size || M >= size) {
            return false;
        }

        // N + CO = K
        if (N + CO != K) {
            return false;
        }

        return true;
    }
};


class Population
{
public:
    // Type Aliases

    // Define Entity & Fitness Pop using SoA (Structure of Arrays)
    typedef thrust::pair<Real, Real> Params;
    typedef thrust::device_vector<Params> EntityPopDevice;
    typedef thrust::device_vector<Real> FitnessPopDevice;
    typedef thrust::host_vector<Params> EntityPopHost;
    typedef thrust::host_vector<Real> FitnessPopHost;


    // Equation :
    //
    // Sin[x - 15] / x * (y - 7) (y - 30) (y - 50) (x - 15) (x - 45)
    //
    // Range : (x, y) in [9, 100] x [7, 50]

public:
    // Public API

    /*!
     * Ctor
     *
     * @param settings settings for the algorithm
     */
    Population(Settings settings)
        : settings(settings) {
    }

    /// Apply the genetic algorithm until the population stabilise and return the best entity
    Params run() {
        // Use a counter for random number so that the random number are really random !
        thrust::counting_iterator<std::size_t> randomCount(0); // (for generator only)

        // Step 1 + 2.
        // -----------
        //
        // Generate a population & evaluate it
        EntityPopDevice epopd(settings.size);
        FitnessPopDevice fpopd(settings.size);
        thrust::transform(randomCount, randomCount + settings.size, epopd.begin(), generator);
        randomCount += settings.size;
        // Evaluate it
        thrust::transform(epopd.begin(), epopd.end(), fpopd.begin(), evaluator);
        // Now sort it
        thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

        // Copy data back to host
        EntityPopHost epoph = epopd;
        FitnessPopHost fpoph = fpopd;

        // Random generators
        thrust::default_random_engine rng;

        unsigned int rounds = 0;

        do {
            ++rounds;

            // Step 3.
            // -------
            //
            // Remove the worse K individuals

            // Skipped -> replace those entities with step 5 & 6


            // Step 4.
            // -------
            //
            // Mutate M individuals of the population

            // Choose M random individuals from the living ones, that is in range [0, size-K[

            for (unsigned int count = 0; count < settings.M; ++count) {
                const unsigned int rangeStart = 0;
                const unsigned int rangeEnd = settings.size - settings.K - 1;
                thrust::uniform_int_distribution<unsigned int> uniform(rangeStart, rangeEnd);
                const unsigned int index = uniform(rng);

                // mutate the entity and recompute its fitness
                epoph[index] = mutator(epoph[index]);
                fpoph[index] = evaluator(epoph[index]);
            }


            // Step 5.
            // -------
            //
            // Create CO new individuals with CrossOver

            // Replace the last CO entities before the N last ones (see comment at step 3)
            for (unsigned int i = settings.size - settings.N - 1, count = 0; count < settings.CO; ++count) {
                // Select two random entities from the living ones, that is in range [0, size-K[
                const unsigned int rangeStart = 0;
                const unsigned int rangeEnd = settings.size - settings.K - 1;
                thrust::uniform_int_distribution<unsigned int> uniform(rangeStart, rangeEnd);
                const unsigned int first = uniform(rng);
                const unsigned int second = uniform(rng);

                epoph[i] = crossover(epoph[first], epoph[second]);
                fpoph[i] = evaluator(epoph[i]);
            }


            // Step 6.
            // -------
            //
            // Generate N new individuals randomly

            // Replace the last N entities (see comment at step 3)
            for (unsigned int i = settings.size - 1, count = 0; count < settings.N; ++count, --i) {
                epoph[i] = generator(*randomCount);
                ++randomCount;
                fpoph[i] = evaluator(epoph[i]);
            }


            // Step 7.
            // -------
            //
            // Evaluate the current population

            // The evaluation of new entities was already done in step 3 to 6
            // So we only sort the population

            // Copy data to device
            epopd = epoph;
            fpopd = fpoph;

            // Sort the data
            thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

            // Copy data back to host
            epoph = epopd;
            fpoph = fpopd;


            // Step 8.
            // -------
            //
            // Goto Step 3 if the population is not stable yet

        } while (!terminator(epoph));

        std::cout << "#rounds = " << rounds << std::endl;

        // Step 9.
        // -------
        //
        // Identify the best individual from the current population

        return epoph.front(); // the population is already sorted;
    }

// private:
    // Private API
    // But public to work with thrust / cuda ...

    static const Real MIN_X = 9, MAX_X = 100, MIN_Y = 7, MAX_Y = 50;

    // Generator; random parameters in [MIN_X, MAX_X] x [MIN_Y, MAX_Y]
    struct Generator {
        Generator()
            : rng(std::time(0))
            , distX(MIN_X, MAX_X)
            , distY(MIN_Y, MAX_Y) {
        }

        __host__ __device__
        Params operator()(std::size_t n) { // The n is used to drop some random numbers
            rng.discard(2 * n); // since we take two random numbers
            return Params(distX(rng), distY(rng));
        }

    private:
        // Random generators
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<Real> distX, distY;
    } generator;

    // Evaluator; the biggest the better
    struct Evaluator {
        __host__ __device__
        Real operator()(Params const& ps) {
            Real x = ps.first, y = ps.second;

            return std::sin(x - 15) / x * (y - 7) * (y - 30) * (y - 50) * (x - 15) * (x - 45);
        }
    } evaluator;

    // CrossOver; takes the average of the two entities
    __host__ __device__
    Params crossover(Params const& as, Params const& bs) {
        Real ax = as.first,
             ay = as.second,
             bx = bs.first,
             by = bs.second;

        return Params((ax + bx) / Real(2), (ay + by) / Real(2));
    }


    // Mutator; takes a normal distribution to shift the current value
    __host__ __device__
    Params mutator(Params const& ps) {
        // TODO implement me !
        return ps;
    }


    // Terminator; stop evolution when population has (relatively) converged
    bool terminator(EntityPopHost const& pop) {
        // Compute average on x and y axes
        Real avgX(0), avgY(0);
        for (EntityPopHost::const_iterator itps = pop.begin(); itps != pop.end(); ++itps) {
            Real x = (*itps).first;
            Real y = (*itps).second;

            avgX += x;
            avgY += y;
        }
        avgX /= pop.size();
        avgY /= pop.size();

        // Stop when 75% of the population is in the range [(1 - ε) * µ, (1 + ε) * µ]
        const unsigned int maxOuts = pop.size() * 0.25;
        const Real EPSILON = 0.02;

        unsigned int outs = 0;
        for (EntityPopHost::const_iterator itps = pop.begin(); itps != pop.end(); ++itps) {
            Real x = (*itps).first;
            Real y = (*itps).second;

            if (!isClose(x, avgX, EPSILON) || !isClose(y, avgY, EPSILON)) {
                ++outs;
            }
        }

        return outs <= maxOuts;
    }

private:
    // Data
    Settings settings;
};


struct Action {
    Action(Population& popref)
        : popref(popref) {
    }

    Population::Params operator()() const {
        return popref.run();
    }

    std::string csvdescription() const {
        return "ø"; // no explicit parameters for the computation
    }

    Population& popref;
};

std::ostream& operator<<(std::ostream& out, Population::Params const& ps)
{
    return out << ps.first << "," << ps.second;
}

#include "stats.hpp"

int main(int, char const**)
{
    // Settings
    const Settings settings(1000, 100, 50, 50, 50);

    // Create the population
    Population pop(settings);

    // Run the Genetic Algorithm
    stats<Action, Population::Params>(Action(pop), 100);

    return 0;
}


