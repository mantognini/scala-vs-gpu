#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>

typedef float Real;

__host__ __device__
bool isClose(Real value, Real target, Real flex)
{
    return (1 - flex) * target <= value && value <= (1 + flex) * target;
}

template <typename T, typename U>
struct SumPair {
    typedef typename thrust::pair<T, U> Pair;

    SumPair() {
    }

    __host__ __device__
    Pair operator()(Pair const& as, Pair const& bs) const {
        return Pair(as.first + bs.first, as.second + bs.second);
    }
};

struct Settings {
    Settings(unsigned int size, unsigned int K)
        : size(size)
        , K(K) {
        if (!isValid()) {
            throw new std::domain_error("Invalid settings");
        }
    }

    const unsigned int size; ///< population size
    const unsigned int K; ///< number of killed per generation

    /// Make sure the settings are valid
    bool isValid() const {
        return K < size;
    }
};


class Population
{
public:
    // Type Aliases

    // Define Entity & Fitness Pop using SoA (Structure of Arrays)
    typedef thrust::pair<Real, Real> Params;
    typedef thrust::device_vector<Params> EntityPopDevice;
    typedef thrust::device_vector<Real> FitnessPopDevice;
    typedef thrust::host_vector<Params> EntityPopHost;
    typedef thrust::host_vector<Real> FitnessPopHost;


    // Equation :
    //
    // Sin[x - 15] / x * (y - 7) (y - 30) (y - 50) (x - 15) (x - 45)
    //
    // Range : (x, y) in [9, 100] x [7, 50]

public:
    // Public API

    /*!
     * Ctor
     *
     * @param settings settings for the algorithm
     */
    Population(Settings settings)
        : settings(settings) {
        std::srand(std::time(0));
    }

    /// Apply the genetic algorithm until the population stabilise and return the best entity
    Params run() {
        // Use a counter for random number so that the random number are really random !
        thrust::counting_iterator<std::size_t> randomCount(0); // (for generator only)

        // And init the random generator of generator
        generator.setSeed(rand());

        // Step 1 + 2.
        // -----------
        //
        // Generate a population & evaluate it
        EntityPopDevice epopd(settings.size);
        FitnessPopDevice fpopd(settings.size);
        thrust::transform(randomCount, randomCount + settings.size, epopd.begin(), generator);
        randomCount += settings.size;
        // Evaluate it
        thrust::transform(epopd.begin(), epopd.end(), fpopd.begin(), evaluator);
        // Now sort it
        thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

        unsigned int rounds = 0;

        do {
            ++rounds;

            // Step 3 + 4
            // ----------
            //
            // Remove the worse K individuals & generate K new individuals randomly

            // Replace the last N entities
            thrust::transform(randomCount, randomCount + settings.K, epopd.end() - settings.K - 1, generator);
            randomCount += settings.K;
            // Evaluate it
            thrust::transform(epopd.end() - settings.K - 1, epopd.end(),
                              fpopd.begin()  - settings.K - 1,
                              evaluator);

            // Step 5.
            // -------
            //
            // Mutate some individuals of the population

            // TODO use prob of mutation instead of fixed settings.
            // use increasing prob of mutation when the entity is far from max


            // Step 6.
            // -------
            //
            // Evaluate the current population

            // The evaluation of new entities was already done in step 3 to 6
            // So we only sort the population

            // Sort the data
            thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());


            // Step 7.
            // -------
            //
            // Goto Step 3 if the population is not stable yet

        } while (!terminator(epopd));

        std::cout << "#rounds = " << rounds << std::endl;

        // Step 8.
        // -------
        //
        // Identify the best individual from the current population

        return epopd.front(); // the population is already sorted;
    }

// private:
    // Private API
    // But public to work with thrust / cuda ...

    static const Real MIN_X = 9, MAX_X = 100, MIN_Y = 7, MAX_Y = 50;

    // Generator; random parameters in [MIN_X, MAX_X] x [MIN_Y, MAX_Y]
    struct Generator {
        Generator()
            : rng(std::time(0))
            , distX(MIN_X, MAX_X)
            , distY(MIN_Y, MAX_Y) {
        }

        void setSeed(unsigned int seed) {
            rng.seed(seed);
        }

        __host__ __device__
        Params operator()(std::size_t n) { // The n is used to drop some random numbers
            rng.discard(2 * n); // since we take two random numbers
            return Params(distX(rng), distY(rng));
        }

    private:
        // Random generators
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<Real> distX, distY;
    } generator;

    // Evaluator; the biggest the better
    struct Evaluator {
        __host__ __device__
        Real operator()(Params const& ps) {
            Real x = ps.first, y = ps.second;

            return std::sin(x - 15) / x * (y - 7) * (y - 30) * (y - 50) * (x - 15) * (x - 45);
        }
    } evaluator;


    // Mutator; takes a normal distribution to shift the current value
    __host__ __device__
    Params mutator(Params const& ps) {
        // TODO implement me !
        return ps;
    }

    struct IsOut {
        IsOut(Real avgX, Real avgY, Real epsilon)
            : avgX(avgX)
            , avgY(avgY)
            , epsilon(epsilon) {
        }

        __host__ __device__
        bool operator()(Params const& ps) const {
            return !isClose(ps.first, avgX, epsilon) || !isClose(ps.second, avgY, epsilon);
        }

        const Real avgX, avgY, epsilon;
    };

    // Terminator; stop evolution when population has (relatively) converged
    bool terminator(EntityPopDevice const& pop) {
        // Compute average on x and y axes
        const SumPair<Real, Real> reducer;
        Params sum = thrust::reduce(pop.begin(), pop.end(), Params(0, 0), reducer);
        Real avgX = sum.first / pop.size();
        Real avgY = sum.second / pop.size();

        // Stop when 75% of the population is in the range [(1 - ε) * µ, (1 + ε) * µ]
        const std::size_t maxOuts = pop.size() * 0.25;
        const Real EPSILON = 0.02;

        const IsOut predicate(avgX, avgY, EPSILON);
        const std::size_t outs = thrust::count_if(pop.begin(), pop.end(), predicate);

        return outs <= maxOuts;
    }

private:
    // Data
    Settings settings;
};


struct Action {
    Action(Population& popref)
        : popref(popref) {
    }

    Population::Params operator()() const {
        return popref.run();
    }

    std::string csvdescription() const {
        return "ø"; // no explicit parameters for the computation
    }

    Population& popref;
};

std::ostream& operator<<(std::ostream& out, Population::Params const& ps)
{
    return out << ps.first << "," << ps.second;
}

#include "stats.hpp"

int main(int, char const**)
{
    // Settings
    const Settings settings(1000, 100);

    // Create the population
    Population pop(settings);

    // Run the Genetic Algorithm
    stats<Action, Population::Params>(Action(pop), 100);

    return 0;
}


