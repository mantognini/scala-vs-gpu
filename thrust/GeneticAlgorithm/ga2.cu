#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/count.h>

#warning THIS IMPLEMENTATION IS IMPRECISE

typedef float Real;

__host__ __device__
bool isClose(Real value, Real target, Real flex)
{
    return (1 - flex) * target <= value && value <= (1 + flex) * target;
}

__host__ __device__
Real clamp(Real value, Real min, Real max)
{
    const Real diff = max - min;
    while (value < min) {
        value += diff;
    }

    while (value > max) {
        value -= diff;
    }

    return value;
}

template <typename T, typename U>
struct SumPair {
    typedef typename thrust::pair<T, U> Pair;

    SumPair() {
    }

    __host__ __device__
    Pair operator()(Pair const& as, Pair const& bs) const {
        return Pair(as.first + bs.first, as.second + bs.second);
    }
};

struct Settings {
    Settings(unsigned int size, unsigned int K)
        : size(size)
        , K(K) {
        if (!isValid()) {
            throw new std::domain_error("Invalid settings");
        }
    }

    const unsigned int size; ///< population size
    const unsigned int K; ///< number of killed per generation

    /// Make sure the settings are valid
    bool isValid() const {
        return K < size;
    }
};


class Population
{
public:
    // Type Aliases

    // Define Entity & Fitness Pop using SoA (Structure of Arrays)
    typedef thrust::pair<Real, Real> Params;
    typedef thrust::device_vector<Params> EntityPopDevice;
    typedef thrust::device_vector<Real> FitnessPopDevice;
    typedef thrust::host_vector<Params> EntityPopHost;
    typedef thrust::host_vector<Real> FitnessPopHost;


    // Equation :
    //
    // Sin[x - 15] / x * (y - 7) (y - 30) (y - 50) (x - 15) (x - 45)
    //
    // Range : (x, y) in [9, 100] x [7, 50]

public:
    // Public API

    /*!
     * Ctor
     *
     * @param settings settings for the algorithm
     */
    Population(Settings settings)
        : settings(settings) {
        std::srand(std::time(0));
    }

    /// Apply the genetic algorithm until the population stabilise and return the best entity
    Params run() {
        // Use a counter for random number so that the random number are really random !
        thrust::counting_iterator<std::size_t> randomCount(0); // (for generator only)

        // And init the random generator of generator and mutator
        generator.setSeed(rand());
        mutator.setSeed(rand());

        // Step 1 + 2.
        // -----------
        //
        // Generate a population & evaluate it
        EntityPopDevice epopd(settings.size);
        FitnessPopDevice fpopd(settings.size);
        thrust::transform(randomCount, randomCount + settings.size, epopd.begin(), generator);
        randomCount += settings.size;
        // Evaluate it
        thrust::transform(epopd.begin(), epopd.end(), fpopd.begin(), evaluator);
        // Now sort it
        thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

        unsigned int rounds = 0;

        do {
            ++rounds;

            // Step 3 + 4
            // ----------
            //
            // Remove the worse K individuals & generate K new individuals randomly

            // Replace the last N entities
            thrust::transform(randomCount, randomCount + settings.K, epopd.end() - settings.K - 1, generator);
            randomCount += settings.K;
            // Evaluate tehm
            thrust::transform(
                epopd.end() - settings.K - 1, epopd.end(), // input
                fpopd.begin()  - settings.K - 1,           // ouput
                evaluator                                  // mapper
            );
            thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());

            // Step 5.
            // -------
            //
            // Mutate some individuals of the population

            // Use prob of mutation instead of fixed settings (if close to max, then probably not mutated)
            mutator.maxfitness = fpopd.front();
            mutator.best = epopd.front();
            thrust::transform_if(
                thrust::make_zip_iterator(                  // data input start
                    thrust::make_tuple(
                        epopd.begin(),                              // actual data
                        thrust::counting_iterator<std::size_t>(0)   // random 'index'
                    )
                ),
                thrust::make_zip_iterator(                  // data input end
                    thrust::make_tuple(
                        epopd.end(),
                        thrust::counting_iterator<std::size_t>(epopd.size())
                    )
                ),
                fpopd.begin(),                  // controller input
                epopd.begin(),                  // data output (in-place)
                mutator,                        // mapper             [ operator(Params) ]
                mutator                         // controller         [ operator(Real)   ]
            );

            // Step 6.
            // -------
            //
            // Evaluate the current population

            // The evaluation of new entities was already done in step 3 to 6
            // So we only sort the population

            // Sort the data
            thrust::sort_by_key(fpopd.begin(), fpopd.end(), epopd.begin());


            // Step 7.
            // -------
            //
            // Goto Step 3 if the population is not stable yet

        } while (!terminator(epopd) && rounds < 10000);

        std::cout << "#rounds = " << rounds << std::endl;

        // Step 8.
        // -------
        //
        // Identify the best individual from the current population

        return epopd.front(); // the population is already sorted;
    }

// private:
    // Private API
    // But public to work with thrust / cuda ...

    static const Real MIN_X = 9, MAX_X = 100, MIN_Y = 7, MAX_Y = 50;

    // Generator; random parameters in [MIN_X, MAX_X] x [MIN_Y, MAX_Y]
    struct Generator {
        Generator()
            : rng(std::rand())
            , distX(MIN_X, MAX_X)
            , distY(MIN_Y, MAX_Y) {
        }

        void setSeed(unsigned int seed) {
            rng.seed(seed);
        }

        __host__ __device__
        Params operator()(std::size_t n) { // The n is used to drop some random numbers
            rng.discard(2 * n); // since we take two random numbers
            return Params(distX(rng), distY(rng));
        }

    private:
        // Random generators
        thrust::default_random_engine rng;
        thrust::uniform_real_distribution<Real> distX, distY;
    } generator;

    // Evaluator; the biggest the better
    struct Evaluator {
        __host__ __device__
        Real operator()(Params const& ps) {
            Real x = ps.first, y = ps.second;

            return std::sin(x - 15) / x * (y - 7) * (y - 30) * (y - 50) * (x - 15) * (x - 45);
        }
    } evaluator;


    // Mutator; takes a normal distribution to shift the current value
    struct Mutator {
        Mutator()
            : rng(std::rand()) {
        }

        // Mutate action
        __host__ __device__
        Params operator()(thrust::tuple<Params, std::size_t> const& tuple) {
            Params ps = thrust::get<0>(tuple);
            const std::size_t n = thrust::get<1>(tuple);
            rng.discard(2 * n);
            thrust::normal_distribution<Real> distX(best.first, (MAX_X - MIN_X) / 8);
            thrust::normal_distribution<Real> distY(best.second, (MAX_Y - MIN_Y) / 8);
            ps.first = clamp(ps.first + distX(rng), MIN_X, MAX_X);
            ps.second = clamp(ps.second + distY(rng), MIN_Y, MAX_Y);
            return ps;
        }

        // Mutate decider
        __host__ __device__
        bool operator()(Real fitness) {
            return fitness / maxfitness < 0.5;
        }

        void setSeed(unsigned int seed) {
            rng.seed(seed);
        }

        Real maxfitness; // must be updated before calling mutate decider !
        Params best;

    private:
        // Random generators
        thrust::default_random_engine rng;
    } mutator;

    struct IsOut {
        IsOut(Real avgX, Real avgY, Real epsilon)
            : avgX(avgX)
            , avgY(avgY)
            , epsilon(epsilon) {
        }

        __host__ __device__
        bool operator()(Params const& ps) const {
            return !isClose(ps.first, avgX, epsilon) || !isClose(ps.second, avgY, epsilon);
        }

        const Real avgX, avgY, epsilon;
    };

    // Terminator; stop evolution when population has (relatively) converged
    bool terminator(EntityPopDevice const& pop) {
        // Compute average on x and y axes
        const SumPair<Real, Real> reducer;
        Params sum = thrust::reduce(pop.begin(), pop.end(), Params(0, 0), reducer);
        Real avgX = sum.first / pop.size();
        Real avgY = sum.second / pop.size();

        // Stop when P % of the population is in the range [(1 - ε) * µ, (1 + ε) * µ]
        const Real P = 75;
        const std::size_t maxOuts = pop.size() * (Real(1) - P / Real(100));
        const Real EPSILON = 0.05;

        const IsOut predicate(avgX, avgY, EPSILON);
        const std::size_t outs = thrust::count_if(pop.begin(), pop.end(), predicate);

        return outs <= maxOuts;
    }

private:
    // Data
    Settings settings;
};


struct Action {
    Action(Population& popref)
        : popref(popref) {
    }

    Population::Params operator()() const {
        return popref.run();
    }

    std::string csvdescription() const {
        return "ø"; // no explicit parameters for the computation
    }

    Population& popref;
};

std::ostream& operator<<(std::ostream& out, Population::Params const& ps)
{
    return out << ps.first << "," << ps.second;
}

#include "stats.hpp"

int main(int, char const**)
{
    // Settings
    const Settings settings(1000, 100);

    // Create the population
    Population pop(settings);

    // Run the Genetic Algorithm
    stats<Action, Population::Params>(Action(pop), 100);

    return 0;
}


