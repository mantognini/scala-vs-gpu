#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include "cuda_complex.hpp"
#include "stats.hpp"

#ifdef SAVE_IMAGE
    #include <SFML/Graphics.hpp>
#endif

typedef float Real;

typedef complex<Real> Complex;

std::ostream& operator<<(std::ostream& out, Complex const& z)
{
    return out << "(" << z.real() << ";" << z.imag() << ")";
}

typedef std::pair<Complex, Complex> ComplexRange;

std::ostream& operator<<(std::ostream& out, ComplexRange const& range)
{
    return out << "{" << range.first << ";" << range.second << "}";
}

typedef bool Color;

static const Color inSetColor = true;
static const Color notInSetColor = false;

typedef std::size_t Index;

struct Mandelbrot : public thrust::unary_function<Index, Color>
{
    Mandelbrot(std::size_t side, std::size_t maxIterations,
               ComplexRange const& range)
    : side(side)
    , maxIterations(maxIterations)
    , range(range)
    { /* - */ }

    // Perform the set computation
    void operator()() const
    {
        // Create an array on the device
        const std::size_t size = side * side;
        thrust::device_vector<Color> deviceImg(size);

        // Then, transform the indexes into 'colors'
        thrust::transform(thrust::counting_iterator<Index>(0),
                          thrust::counting_iterator<Index>(size),
                          deviceImg.begin(),
                          *this); // apply op()(Index)

        // Copy the data to the host memory
        thrust::host_vector<Color> img = deviceImg;

        #ifdef SAVE_IMAGE
        static std::size_t imgId = 0;

        // Export it to png
        sf::Image png; png.create(side, side, sf::Color::White);
        for (std::size_t x = 0; x < side; ++x) {
            for (std::size_t y = 0; y < side; ++y) {
                png.setPixel(x, y, img[y * side + x] == inSetColor ? sf::Color::Black : sf::Color::White);
            }
        }

        std::stringstream filename;
        filename << "tmp/fractal_"
                 << imgId
                 << "_"
                 << csvdescription()
                 << ".png";
        png.saveToFile(filename.str());
        ++imgId;
        #endif
    }

    __host__ __device__
    Color operator()(Index const& index)
    {
        const unsigned int x = index % side;
        const unsigned int y = index / side; // integer division

        const Complex c(
            range.first.real() + x / (side - Real(1.0f)) * (range.second.real() - range.first.real()),
            range.first.imag() + y / (side - Real(1.0f)) * (range.second.imag() - range.first.imag())
        );

        Complex z( 0, 0 );

        std::size_t iter = 0;
        for (iter = 0; iter < maxIterations && abs(z) < Real(2.0f); ++iter) {
            z = z * z + c;
        }

        return iter == maxIterations ? inSetColor : notInSetColor;
    }

    std::string csvdescription() const 
    {
        std::stringstream ss;
        ss << side << "," 
           << maxIterations << "," 
           << range;
        return ss.str();
    }

    std::size_t side, maxIterations;
    ComplexRange range;
};

int main(int, char**)
{
    const std::size_t sides[] = { 100, 200, 400, 800, 1200, 1600, 2000, 4000, 10000 };
    const std::size_t sidesCount = 9;
    const std::size_t iterations[] = { 1, 10, 30, 80, 150, 250, 500, 1000, 2000, 8000 };
    const std::size_t iterationsCount = 10;
    const ComplexRange ranges[] = {
        ComplexRange( Complex(-1.72, 1.2), Complex(1.0, -1.2) ),
        ComplexRange( Complex(-0.7, 0), Complex(0.3, -1) ),
        ComplexRange( Complex(-0.4, -0.5), Complex(0.1, -1) ),
        ComplexRange( Complex(-0.4, -0.6), Complex(-0.2, -0.8) ),
        ComplexRange( Complex(-0.24, -0.64), Complex(-0.26, -0.66) )
    };
    const std::size_t rangesCount = 5;

    #ifdef SAVE_IMAGE
    const std::size_t repetitions = 1;
    #else
    const std::size_t repetitions = 4;
    #endif

    for (std::size_t s = 0; s < sidesCount; ++s)
        for (std::size_t i = 0; i < iterationsCount; ++i)
            for (std::size_t r = 0; r < rangesCount; ++r)
                stats<Mandelbrot, void>(Mandelbrot(sides[s], iterations[i], ranges[r]), 
                                        iterations[i] >= 1000 && sides[s] >= 2000 ? 1 : repetitions);

    return 0;
}
