#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/sequence.h>

#include <SFML/Graphics.hpp>

#include "cuda_complex.hpp"
typedef complex<double> Complex;
typedef std::pair<Complex, Complex> ComplexRange;

typedef unsigned int Color;
typedef Color Index;
// Note : indexes are stored as color to reduce memory usage (we transform Indexes into Colors in place)

struct Mandelbrot : public thrust::unary_function<Index, Color> {
    std::size_t width, height;
    Color inSetColor, notInSetColor;
    ComplexRange range;
    std::size_t maxIterations;

    Mandelbrot(std::size_t width, std::size_t height, Color in, Color out, ComplexRange range, std::size_t maxIterations)
        : width(width), height(height), inSetColor(in), notInSetColor(out), range(range), maxIterations(maxIterations) {
        /* that's it */
    }

    __host__ __device__
    Color operator()(Index const& index) {
        const unsigned int x = index % width;
        const unsigned int y = index / height; // integer division

        Complex c(
            range.first.real() + x / (width - 1.0) * (range.second.real() - range.first.real()),
            range.first.imag() + y / (width - 1.0) * (range.second.imag() - range.first.imag())
        );

        Complex z( 0, 0 );

        std::size_t iter = 0;
        for (iter = 0; iter < maxIterations && abs(z) < 2.0; ++iter) {
            z = z * z + c;
        }

        return iter == maxIterations ? inSetColor : notInSetColor;
    }
};

int main(int argc, char** argv)
{
    const std::size_t WIDTH = 2000;
    const std::size_t HEIGHT = 2000;
    const Color inSet = 0xffffff;
    const Color notInSet = 0x000000;
    const ComplexRange range ( Complex(-1.72, 1.2), Complex(1.0, -1.2) );
    const std::size_t iterations = 1000;

    // Create an array on the device
    thrust::device_vector<Color> deviceImg(WIDTH * HEIGHT);

    // First, load all indexes into deviceImg
    thrust::sequence(deviceImg.begin(), deviceImg.end());

    // Then, transform the indexes into 'colors'
    thrust::transform(deviceImg.begin(), deviceImg.end(),
                      deviceImg.begin(),
                      Mandelbrot(WIDTH, HEIGHT, inSet, notInSet, range, iterations));

	// Copy the data to the host memory
    thrust::host_vector<Color> img(deviceImg.begin(), deviceImg.end());

    // Export it to png
    sf::Image png; png.create(WIDTH, HEIGHT, sf::Color::White);
    for (std::size_t x = 0; x < WIDTH; ++x) {
        for (std::size_t y = 0; y < HEIGHT; ++y) {
            if (img[y * WIDTH + x] == inSet) png.setPixel(x, y, sf::Color::Black);
        }
    }

    png.saveToFile("fractal.png");

    return 0;
}
