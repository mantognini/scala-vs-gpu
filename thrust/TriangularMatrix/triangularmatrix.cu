
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <sstream>
#include <vector>

typedef float Real;
typedef thrust::host_vector<Real> Matrix;
typedef thrust::device_vector<Real> MatrixOnDevice;
typedef Real* PointerOnDevice;

std::ostream& operator<<(std::ostream& out, Matrix const& m);

#include "stats.hpp"

//
// Compute the NxN matrix multiplication of a lower triangular matrix A with a square matrix B
//

// Disable output of matrix data
#define OUTPUT_MATRIX_DATA

std::ostream& operator<<(std::ostream& out, Matrix const& m)
{
#ifdef OUTPUT_MATRIX_DATA
    const std::size_t NN = m.size();
    for (std::size_t i = 0; i < NN; ++i) {
        out << m[i] << (i < NN - 1 ? ";" : "");
    }
    return out;
#else
    return out << "skipped";
#endif
}

struct TriMatrixMul
{
    // N is the dimension of the matrixes
    TriMatrixMul(std::size_t N)
    : N(N)
    { /* */ }

    struct Computer
    {
        Computer(PointerOnDevice const& A, PointerOnDevice const& B, std::size_t N)
        : A(B)
        , B(B)
        , N(N)
        { /*  */ }

        __host__ __device__
        Real operator()(std::size_t ij)
        {
            // Unmap ij to (i, j)
            const std::size_t i = ij % N;
            const std::size_t j = ij / N;

            // Compute the offset for A
            std::size_t offset = 0;
            for (std::size_t x = 0; x <= i; offset += ++x);

            // Compute the element
            Real sum = 0;
            for (std::size_t k = 0; k <= i; ++k) {
                sum += A[offset + k] * B[k * N + j];
            }
            
            return sum;
        }

        PointerOnDevice const& A;
        PointerOnDevice const& B;
        std::size_t N;
    };

    // Perform the computation
    Matrix operator()() const {
        // Create A, a lower triangular matrix

        // Memory allocation
        PointerOnDevice dARawPtr = 0;
        const std::size_t dASize = N * (N + 1) / 2;
        hipMalloc((void**)&dARawPtr, dASize * sizeof(Real));

        // Init its values with 0, 1, 2, ... M
        thrust::device_ptr<Real> dAPtr = thrust::device_pointer_cast(dARawPtr);
        thrust::sequence(dAPtr, dAPtr + dASize, 0.0f);

        // Create B, a square matrix

        // Memory allocation
        PointerOnDevice dBRawPtr = 0;
        const std::size_t dBSize = N * N;
        hipMalloc((void**)&dBRawPtr, dBSize * sizeof(Real));

        // Init its values with 0, 1, 2, ... N * N
        thrust::device_ptr<Real> dBPtr = thrust::device_pointer_cast(dBRawPtr);
        thrust::sequence(dBPtr, dBPtr + dBSize, 0.0f);

        // Create result matrix C

        // Create C, a square matrix for the result of A * B

        // Memory allocation
        PointerOnDevice dCRawPtr = 0;
        const std::size_t dCSize = N * N;
        hipMalloc((void**)&dCRawPtr, dCSize * sizeof(Real));

        thrust::device_ptr<Real> dCPtr = thrust::device_pointer_cast(dCRawPtr);

        // To perform the computation on the GPU we map (i, j) to ij

        // Launch the kernels
        thrust::counting_iterator<std::size_t> indexesBegin(0);
        thrust::counting_iterator<std::size_t> indexesEnd(N * N);
        Computer computer(dARawPtr, dBRawPtr, N);
        thrust::transform(indexesBegin, indexesEnd, dCPtr, computer);

        // Copy result to the host
        Matrix C(dCSize);
        hipMemcpy(C.data(), dCRawPtr, dCSize * sizeof(Real), hipMemcpyDeviceToHost);

        hipFree(dCRawPtr);
        dCRawPtr = 0;

        hipFree(dBRawPtr);
        dBRawPtr = 0;

        hipFree(dARawPtr);
        dARawPtr = 0;

        return C;
    }

    std::string csvdescription() const {
        std::stringstream ss;
        ss << N;
        return ss.str();
    }

    std::size_t N;
};


int main(int argc, const char * argv[])
{
    // Make stats from N = 2 to N = 2^12
    for (std::size_t N = 2; N <= 4096; N *= 2) {
        stats<TriMatrixMul, Matrix>(TriMatrixMul(N), 4);
    }

    return 0;
}

