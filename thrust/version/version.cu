#include <thrust/version.h>
#include <iostream>

int main(void)
{
  int major = THRUST_MAJOR_VERSION;
  int minor = THRUST_MINOR_VERSION;

  std::cout << "Thrust v" << major << "." << minor << std::endl;

  return 0;
}